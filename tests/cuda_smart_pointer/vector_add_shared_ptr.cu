#include <hip/hip_runtime.h>
#include <iostream>
#include <memory>

struct CudaDeleter {
    void operator()(float *ptr) const {
        hipFree(ptr);
    }
};

// CUDA kernel for vector addition
__global__ void vectorAdd(const float* A, const float* B, float* C, int N) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        C[idx] = A[idx] + B[idx];
    }
}

int main() {
    int N = 10;
    size_t size = N * sizeof(float);

    // Allocate host memory
    float h_A[N], h_B[N], h_C[N];
    for (int i = 0; i < N; ++i) {
        h_A[i] = static_cast<float>(i);
        h_B[i] = static_cast<float>(i * 2);
    }

    // Allocate device memory
    float* d_A;
    float* d_B;
    float* d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // Create unique_ptrs with the custom deleter
    std::unique_ptr<float, CudaDeleter> deviceA(d_A);
    std::unique_ptr<float, CudaDeleter> deviceB(d_B);
    std::unique_ptr<float, CudaDeleter> deviceC(d_C);

    // Copy input data to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Launch the CUDA kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Copy the result back to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Print the result
    std::cout << "Result of vector addition:\n";
    for (int i = 0; i < N; ++i) {
        std::cout << h_C[i] << " ";
    }
    std::cout << std::endl;

    // Device memory will be automatically freed when unique_ptrs go out of scope

    return 0;
}
